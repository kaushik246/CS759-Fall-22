#include "hip/hip_runtime.h"
#include "nlm_sm.cuh"
#include <stdio.h>
#include "utils.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define FILTER_SIGMA 0.0185
__device__ const float DEV_FILTER_SIGMA = (float)FILTER_SIGMA;

__device__ void compare_patches(float *comp_value, float *patch_i, int j, float *G, float *shared_memory, int pixels, int padding, int patch)
{
    int offset = padding * (pixels + 2 * padding);
    j += offset;
    for (int it1 = 0; it1 < patch; it1++)
    {
        for (int it2 = 0; it2 < patch; it2++)
        {
            int first_index = it1 * patch + it2;
            int second_index = j + (it1 - padding) * (pixels + 2 * padding) + it2 - padding;
            if (patch_i[first_index] != (float)-1 && shared_memory[second_index] != (float)-1)
            {
                float diff = patch_i[first_index] - shared_memory[second_index];
                *comp_value += G[first_index] * (diff * diff);
            }
        }
    }
}

__global__ void nlm_sm(float *nlm_image, float *image, int size_with_padding, float *gaussian_arr, int pixels, int padding, int patch)
{
    int index = blockIdx.x * (blockDim.x + 2 * padding) + (threadIdx.x + padding) + padding * pixels + 2 * padding * padding;
    int row_size = pixels + 2 * padding;
    if (index < size_with_padding)
    {
        extern __shared__ float shared_memory[];
        for (int i = 0; i < patch; i++)
        {
            shared_memory[(threadIdx.x + padding) + i * row_size] = image[(threadIdx.x + padding) + i * row_size];
        }
        if (threadIdx.x == 0)
        {
            for (int row = 0; row < padding; row++)
            {
                for (int col = 0; col < patch; col++)
                {
                    shared_memory[row + col * row_size] = -1;
                }
            }
            for (int row = (padding + pixels); row < row_size; row++)
            {
                for (int col = 0; col < patch; col++)
                {
                    shared_memory[row + col * row_size] = -1;
                }
            }
        }
        __syncthreads();
        for (int i = 0; i < patch * padding; i++)
        {
            printf("%f\n", shared_memory[i]);
        }
        float patch_i[9];

        for (int it1 = 0; it1 < patch; it1++)
        {
            for (int it2 = 0; it2 < patch; it2++)
            {
                patch_i[it1 * patch + it2] = image[index + (it1 - padding) * row_size + it2 - padding];
            }
        }

        nlm_image[index] = 0;
        float weight;
        float Z = 0;
        for (int it1 = padding; it1 < pixels + padding; it1++)
        {
            for (int it2 = padding; it2 < (pixels + padding); it2++)
            {
                float comp_value = 0;
                compare_patches(&comp_value, patch_i, it2, gaussian_arr, shared_memory, pixels, padding, patch);
                weight = (float)(exp(-comp_value / (DEV_FILTER_SIGMA * DEV_FILTER_SIGMA)));
                nlm_image[index] += weight * shared_memory[padding * row_size + it2];
                Z += weight;
            }

            __syncthreads();

            for (int i = 0; i < patch - 1; i++)
            {
                shared_memory[(threadIdx.x + padding) + i * row_size] = shared_memory[(threadIdx.x + padding) + (i + 1) * row_size];
            }
            int row_offset = (it1 + 1 - padding) * row_size;

            shared_memory[(threadIdx.x + padding) + (patch - 1) * row_size] = image[row_offset + (threadIdx.x + padding) + (patch - 1) * row_size];
            __syncthreads();
        }

        nlm_image[index] = nlm_image[index] / Z;
    }
}
