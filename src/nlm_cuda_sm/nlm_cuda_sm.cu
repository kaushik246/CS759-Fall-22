#include "nlm_sm.cuh"
#include <stdio.h>
#include <iostream>
#include "utils.h"
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

void printer(float *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        cout << arr[i] << endl;
    }
}

void nlm_cuda(float *image, float *nlm_image, int pixels, int padding, int patch)
{
    int size_with_padding = pixels * pixels + 4 * padding * pixels + 4 * padding * padding;

    float *gaussian_arr;
    gaussian_arr = gaussian_filter(patch);

    float *dev_gaussian_arr;
    hipMallocManaged(&dev_gaussian_arr, patch * patch * sizeof(float));

    for (int i = 0; i < patch * patch; i++)
    {
        dev_gaussian_arr[i] = gaussian_arr[i];
    }

    int shared_memory_size = patch * (pixels + 2 * padding);

    nlm_sm<<<pixels, pixels, shared_memory_size>>>(nlm_image, image, size_with_padding, dev_gaussian_arr, pixels, padding, patch);
    hipDeviceSynchronize();

    free(gaussian_arr);
    hipFree(dev_gaussian_arr);
}

int main(int argc, char *argv[])
{
    int pixels = atoi(argv[1]);
    int patch = atoi(argv[2]);
    string file_name = argv[3];

    int padding = patch / 2;

    int size_with_padding = pixels * pixels + 4 * padding * pixels + 4 * padding * padding;

    float *image;
    image = parse_vals_from_txt(pixels, padding);

    float *dev_image;
    hipMallocManaged((void **)&dev_image, size_with_padding * sizeof(float));

    for (int i = 0; i < size_with_padding; i++)
    {
        dev_image[i] = image[i];
    }

    float *nlm_image;
    hipMallocManaged((void **)&nlm_image, size_with_padding * sizeof(float));
    for (int i = 0; i < size_with_padding; i++)
    {
        nlm_image[i] = (float)-1;
    }

    nlm_cuda(dev_image, nlm_image, pixels, padding, patch);

    write_vals_to_txt(nlm_image, pixels, padding, patch);

    hipFree(nlm_image);
    hipFree(dev_image);
    free(image);

    return 0;
}