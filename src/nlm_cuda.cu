#include "nlm.cuh"
#include <stdio.h>
#include <iostream>
#include "utils.h"
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

void nlm_cuda(float *image, float *nlm_image, int pixels, int padding, int patch)
{
    int size_with_padding = pixels * pixels + 4 * padding * pixels + 4 * padding * padding;

    float *gaussian_arr;
    gaussian_arr = gaussian_filter(patch);

    float *dev_gaussian_arr;
    hipMallocManaged(&dev_gaussian_arr, patch * patch * sizeof(float));

    hipMemcpy(&dev_gaussian_arr, gaussian_arr, sizeof(float) * pixels * pixels, hipMemcpyHostToDevice);

    nlm<<<pixels, pixels>>>(nlm_image, image, size_with_padding, dev_gaussian_arr);
    hipDeviceSynchronize();

    free(gaussian_arr);
    hipFree(dev_gaussian_arr);
}

int main(int argc, char *argv[])
{
    int pixels = atoi(argv[1]);
    int patch = atoi(argv[2]);
    string file_name = argv[3];

    int padding = patch / 2;

    int size_with_padding = pixels * pixels + 4 * padding * pixels + 4 * padding * padding;

    float *image;
    image = parse_vals_from_txt(pixels, padding);

    float *dev_image;
    hipMallocManaged((void **)&dev_image, size_with_padding * sizeof(float));
    hipMemcpy(&dev_image, image, sizeof(float) * size_with_padding, hipMemcpyHostToDevice);

    float *nlm_image;
    hipMallocManaged((void **)&nlm_image, size_with_padding * sizeof(float));
    for (int i = 0; i < size_with_padding; i++)
    {
        filtered_image[i] = (float)-1;
    }

    nlm_cuda(dev_image, nlm_image, pixels, padding, patch);

    write_vals_to_txt(nlm_image, pixels, padding, patch);

    hipFree(nlm_image);
    hipFree(dev_image);
    free(image);

    return 0;
}
