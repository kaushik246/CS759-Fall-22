#include "hip/hip_runtime.h"
#include "nlm.cuh"
#include <stdio.h>
#include <math.h>

#define FILTER_SIGMA 0.0185
#define PATCH_SIGMA 3.1550

__device__ const float DEV_FILTER_SIGMA = (float)FILTER_SIGMA;

__device__ void compare_patches(float *comp, float *image, float *gaussian_arr, int i, int j, int pixels, int padding, int patch)
{
    for (int m = 0; m < patch; m++)
    {
        for (int n = 0; n < patch; n++)
        {
            int idx_1 = i + (m - padding) * (pixels + 2 * padding) + n - padding;
            int idx_2 = j + (m - padding) * (pixels + 2 * padding) + n - padding;
            if (image[idx_1] != (float)-1 && image[idx_2] != (float)-1)
            {
                float diff = image[idx_1] - image[idx_2];
                *comp += gaussian_arr[m * patch + n] * (diff * diff);
            }
        }
    }
}
__global__ void nlm(float *nlm_image, float *image, int size_with_padding, float *gaussian_arr, int pixels, int padding, int patch)
{
    int index = blockIdx.x * (blockDim.x + 2 * padding) + (threadIdx.x + padding) + padding * pixels + 2 * padding * padding;
    int row_size = pixels + 2 * padding;
    if (index < size_with_padding)
    {
        nlm_image[index] = 0;
        float weight;
        float Z = 0;
        for (int i = padding; i < (pixels + padding); i++)
        {
            for (int j = padding; j < (pixels + padding); j++)
            {
                float comp = 0;
                compare_patches(&comp, image, gaussian_arr, index, i * (pixels + 2 * padding) + j, pixels, padding, patch);
                weight = (float)(exp(-comp / (DEV_FILTER_SIGMA * DEV_FILTER_SIGMA)));
                nlm_image[index] += weight * image[i * row_size + j];
                Z += weight;
            }
        }
        nlm_image[index] = nlm_image[index] / Z;
    }
}
