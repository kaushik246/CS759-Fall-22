#include "knn.cuh"
#include <stdio.h>
#include <iostream>
#include "utils.h"
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;

void printer(float *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        cout << arr[i] << endl;
    }
}

void knn_cuda(float *image, float *knn_image, int pixels, int padding, int patch)
{
    int size_with_padding = pixels * pixels + 4 * padding * pixels + 4 * padding * padding;

    float *gaussian_arr;
    gaussian_arr = gaussian_filter(patch);

    float *dev_gaussian_arr;
    hipMallocManaged(&dev_gaussian_arr, patch * patch * sizeof(float));

    for (int i = 0; i < patch * patch; i++)
    {
        dev_gaussian_arr[i] = gaussian_arr[i];
    }

    int shared_memory_size = patch * (pixels + 2 * padding) * sizeof(float);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    knn_sm<<<pixels, pixels, shared_memory_size>>>(knn_image, image, size_with_padding, dev_gaussian_arr, pixels, padding, patch);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);

    free(gaussian_arr);
    hipFree(dev_gaussian_arr);
}

int main(int argc, char *argv[])
{
    int pixels = atoi(argv[1]);
    int patch = atoi(argv[2]);
    string file_name = argv[3];

    int padding = patch / 2;

    int size_with_padding = pixels * pixels + 4 * padding * pixels + 4 * padding * padding;

    float *image;
    image = parse_vals_from_txt(pixels, padding);

    float *dev_image;
    hipMallocManaged((void **)&dev_image, size_with_padding * sizeof(float));

    for (int i = 0; i < size_with_padding; i++)
    {
        dev_image[i] = image[i];
    }

    float *knn_image;
    hipMallocManaged((void **)&knn_image, size_with_padding * sizeof(float));
    for (int i = 0; i < size_with_padding; i++)
    {
        knn_image[i] = (float)-1;
    }

    knn_cuda(dev_image, knn_image, pixels, padding, patch);

    write_vals_to_txt(knn_image, pixels, padding, patch);

    hipFree(knn_image);
    hipFree(dev_image);
    free(image);

    return 0;
}