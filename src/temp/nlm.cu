#include "hip/hip_runtime.h"
#include "nlm.cuh"
#include <stdio.h>
#include "utils.h"
#include <math.h>

__device__ void compare_patches(float *comp, float *image, float *gaussian_arr, int i, int j, int pixels, int padding, int patch)
{
    for (int m = 0; m < patch; m++)
    {
        for (int n = 0; n < patch; n++)
        {
            int idx_1 = i + (m - padding) * (pixels + 2 * padding) + n - padding;
            int idx_2 = j + (m - padding) * (pixels + 2 * padding) + n - padding;
            if (image[idx_1] != (float)-1 && image[idx_2] != (float)-1)
            {
                float diff = image[idx_1] - image[idx_2];
                *comp += gaussian_arr[m * patch + n] * (diff * diff);
            }
        }
    }
}
__global__ void nlm(float *nlm_image, float *image, int size_with_padding, float *gaussian_arr, int pixels, int padding, int patch)
{
    int index = blockIdx.x * (blockDim.x + 2 * padding) + (threadIdx.x + padding) + padding * pixels + 2 * padding * padding;
    int row_size = pixels + 2 * padding;
    if (index < size_with_padding)
    {
        nlm_image[index] = 0;
        float weight;
        float Z = 0;
        for (int i = padding; i < (pixels + padding); i++)
        {
            for (int j = padding; j < (pixels + padding); j++)
            {
                float comp = 0;
                compare_patches(&comp, image, index, i * (pixels + 2 * padding) + j, gaussian_arr, pixels, padding, patch);
                weight = (float)(exp(-comp / (FILTER_SIGMA * FILTER_SIGMA)));
                nlm_image[index] += weight * image[i * row_size + j];
                Z += weight;
            }
        }
        nlm_image[index] = nlm_image[index] / Z;
    }
}
